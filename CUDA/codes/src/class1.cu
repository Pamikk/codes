#include "class2.cuh"
void dataset::test_gpu(int n){
    hipMalloc(&dev_data,sizeof(int)*n);
    fill_dataset<<<10,10>>>(*this);
    hipDeviceSynchronize();
}
void dataset::test_cpu(int n){
    host_data = new int[n];
    printf("????\n");
    fill_dataset(*this,20);
}