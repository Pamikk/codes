#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void gen_tensor(float** tensor){
  float x = blockIdx.x+1;
  float y = threadIdx.x+1;
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  tensor[blockIdx][threadIdx] = sqrt(x*y);
}
 int main()
 { 
   const int n =2;
   const int dim = 3;
   float tensor[n][dim];
   //hipMalloc(&tensor,sizeof(float)*(n*dim));
   gen_tensor<<<dim,n>>>(tensor);
   hipDeviceSynchronize();
   float htensor = new float[n][dim];
   hipMemcpy(htensor,tensor, sizeof(float)*(n*dim),hipMemcpyDeviceToHost);
   for (int i=0;i<dim*n;i++){
      printf("%1.4f\n",htensor[i]);
   } 
 }
 